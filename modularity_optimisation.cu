#include "hip/hip_runtime.h"
#include "modularity_optimisation.cuh"
#include <climits>
#include <cstdio>
#include <thrust/partition.h>

/**
 * Computes hashing (using double hashing) for open-addressing purposes of arrays in prepareHashArrays function.
 * @param val   value we want to insert
 * @param index current position
 * @param prime size of hash array
 * @return hash
 */
__device__ int getHash(int val, int index, int prime) {
	int h1 = val % HASHING;
	int h2 = 1 + (val % (HASHING - 1));
	return (h1 + index * h2) % prime;
}

__device__ float atomicMaxFloat(float *addr, float value) {
	float old;
	old = (value >= 0) ? __int_as_float(atomicMax((int *)addr, __float_as_int(value))) :
		  __uint_as_float(atomicMin((unsigned int *)addr, __float_as_uint(value)));

	return old;
}

/**
 * Computes sum of weights of edges adjacent to vertices (results are stored in vertexEdgesSum).
 * @param deviceStructures structures stored in device memory
 */
__global__ void computeEdgesSum(device_structures deviceStructures) {
	int verticesPerBlock = blockDim.y;
	int concurrentNeighbours = blockDim.x;
	float edgesSum = 0;
	int vertex = blockIdx.x * verticesPerBlock + threadIdx.y;
	if (vertex < *deviceStructures.V) {
		int startOffset = deviceStructures.edgesIndex[vertex], endOffset = deviceStructures.edgesIndex[vertex + 1];
		for (int index = startOffset + threadIdx.x; index < endOffset; index += concurrentNeighbours)
			edgesSum += deviceStructures.weights[index];

		for (unsigned int offset = concurrentNeighbours / 2; offset > 0; offset /= 2) {
			edgesSum += __shfl_down_sync(FULL_MASK, edgesSum, offset);
		}
		if (threadIdx.x == 0) {
			deviceStructures.vertexEdgesSum[vertex] = edgesSum;
		}
	}
}

/**
 * Computes sum of weights of edges adjacent to vertices (results are stored in vertexEdgesSum).
 * @param V               number of vertices
 * @param communityWeight community -> weight (sum of edges adjacent to vertices of community)
 * @param vertexCommunity vertex -> community assignment
 * @param vertexEdgesSum  vertex -> sum of edges adjacent to vertex
 */
__global__ void computeCommunityWeight(device_structures deviceStructures) {
	int vertex = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
	if (vertex < *deviceStructures.V) {
		int community = deviceStructures.vertexCommunity[vertex];
		atomicAdd(&deviceStructures.communityWeight[community], deviceStructures.vertexEdgesSum[vertex]);
	}
}

/**
 * Fills content of hashCommunity and hash_weights arrays that are later used in computeGain function.
 * @param community        neighbour's community
 * @param prime            prime number used for hashing
 * @param weight		   neighbour's weight
 * @param hashWeight	   table of sum of weights between vertices and communities
 * @param hashCommunity	   table informing which community's info is stored in given index
 * @param hashTablesOffset offset of the vertex in hash arrays (single hash array may contain multiple vertices)
 */
__device__ void prepareHashArrays(int community, int prime, float weight, float *hashWeight, int *hashCommunity,
								  int hashTablesOffset) {
	bool foundPosition = false;
	int it = 0;
	while (!foundPosition) {
		int curPos = hashTablesOffset + getHash(community, it++, prime);
		if (hashCommunity[curPos] == community)
			atomicAdd(&hashWeight[curPos], weight);
			// TODO - uses inelegant solution with -1
		else if (hashCommunity[curPos] == -1) {
			if (atomicCAS(&hashCommunity[curPos], -1, community) == -1)
				atomicAdd(&hashWeight[curPos], weight);
			else if (hashCommunity[curPos] == community)
				atomicAdd(&hashWeight[curPos], weight);
		}
		foundPosition = hashCommunity[curPos] == community;
	}
}

/**
 * Computes gain that would be obtained if we would move vertex to community.
 * @param vertex      	   vertex number
 * @param prime            prime number used for hashing (and size of vertex's area in hash arrays)
 * @param community 	   neighbour's community
 * @param currentCommunity current community of vertex
 * @param communityWeight  community -> weight (sum of edges adjacent to vertices of community)
 * @param vertexEdgesSum   vertex -> sum of edges adjacent to vertex
 * @param hashCommunity    table informing which community's info is stored in given index
 * @param hashWeight       table of sum of weights between vertices and communities
 * @param hashTablesOffset offset of the vertex in hash arrays (single hash array may contain multiple vertices
 * @return gain that would be obtained by moving vertex to community
 */
__device__ float computeGain(int vertex, int prime, int community, int currentCommunity, float *communityWeight,
							 float *vertexEdgesSum, int *hashCommunity, float *hashWeight, int hashTablesOffset, int neighbour) {
	float communitySum = communityWeight[community];
	float currentCommunitySum = communityWeight[currentCommunity] - vertexEdgesSum[vertex];
	float vertexToCommunity = 0, vertexToCurrentCommunity = 0;
	for (int i = 0; i < prime; i++) {
		int index = hashTablesOffset + i;
		if (hashCommunity[index] == community)
			vertexToCommunity = hashWeight[index];
		else if (hashCommunity[index] == currentCommunity)
			vertexToCurrentCommunity = hashWeight[index];
	}
	float gain = (vertexToCommunity - vertexToCurrentCommunity) / M +
				 vertexEdgesSum[vertex] * (currentCommunitySum - communitySum) / (2 * M * M);
	return gain;
}

/**
 * Finds new vertex -> community assignment (stored in newVertexCommunity) that maximise gains for each vertex.
 * @param V                number of vertices
 * @param vertices		   vertices
 * @param prime            prime number used for hashing
 * @param deviceStructures structures kept in device memory
 */
__global__ void computeMove(int V, int *vertices, int prime, device_structures deviceStructures) {
	int *vertexCommunity = deviceStructures.vertexCommunity, *edgesIndex = deviceStructures.edgesIndex,
	*edges = deviceStructures.edges, *communitySize = deviceStructures.communitySize,
	*newVertexCommunity = deviceStructures.newVertexCommunity;
	float *weights = deviceStructures.weights, *communityWeight = deviceStructures.communityWeight,
	*vertexEdgesSum = deviceStructures.vertexEdgesSum;

	int verticesPerBlock = blockDim.y;
	int concurrentNeighbours = blockDim.x;
	int hashTablesOffset = threadIdx.y * prime;
	int bestGainsIndex = threadIdx.y;
	int vertexIndex = blockIdx.x * verticesPerBlock + threadIdx.y;

	if (vertexIndex < V) {
		extern __shared__ int s[];
		int *hashCommunity = s;
		auto *hashWeight = (float*)&hashCommunity[verticesPerBlock * prime];
		for (int i = 0; i < prime; i++) {
			hashWeight[hashTablesOffset + i] = 0;
			hashCommunity[hashTablesOffset + i] = -1;
		}

		int vertex = vertices[vertexIndex];
		int currentCommunity = vertexCommunity[vertex];
		int bestCommunity = currentCommunity;
		float bestGain = 0;
		// putting data in hash table
		int neighbourIndex = threadIdx.x + edgesIndex[vertex];

		while (neighbourIndex < edgesIndex[vertex + 1]) {
			int neighbour = edges[neighbourIndex];
			int community = vertexCommunity[neighbour];
			float weight = weights[neighbourIndex];
			// this lets us achieve ei -> C(i)\{i} instead of ei -> C(i)
			if (neighbour != vertex)
				prepareHashArrays(community, prime, weight, hashWeight, hashCommunity, hashTablesOffset);
			neighbourIndex += concurrentNeighbours;
		}

		if (concurrentNeighbours > WARP_SIZE)
			__syncthreads();

		// choosing community
		neighbourIndex = threadIdx.x + edgesIndex[vertex];
		while (neighbourIndex < edgesIndex[vertex + 1]) {
			int neighbour = edges[neighbourIndex];
			int community = vertexCommunity[neighbour];
			// TODO - should we check `community != currentCommunity` for sure?
			if ((community < currentCommunity || communitySize[community] > 1 || communitySize[currentCommunity] > 1) &&
			community != currentCommunity) {
				float gain = computeGain(vertex, prime, community, currentCommunity, communityWeight, vertexEdgesSum,
										 hashCommunity, hashWeight, hashTablesOffset, neighbour);
				if (gain > bestGain || (gain == bestGain && community < bestCommunity)) {
					bestGain = gain;
					bestCommunity = community;
				}
			}
			neighbourIndex += concurrentNeighbours;
		}

		if (concurrentNeighbours > WARP_SIZE)
			__syncthreads();

		if (concurrentNeighbours <= WARP_SIZE) {
			for (unsigned int offset = concurrentNeighbours / 2; offset > 0; offset /= 2) {
				float otherGain = __shfl_down_sync(FULL_MASK, bestGain, offset);
				int otherCommunity = __shfl_down_sync(FULL_MASK, bestCommunity, offset);
				if (otherGain > bestGain || (otherGain == bestGain && otherCommunity < bestCommunity)) {
					bestGain = otherGain;
					bestCommunity = otherCommunity;
				}
			}
			if (threadIdx.x == 0) {
				newVertexCommunity[vertex] = bestCommunity;
			}
		} else {

		}
	}
}

/**
 * Updates vertexCommunity content based on newVertexCommunity content..
 * Additionally, updates communitySize.
 * @param V                number of vertices
 * @param vertices         vertices
 * @param deviceStructures structures kept in device memory
 */
__global__ void updateVertexCommunity(int V, int *vertices, device_structures deviceStructures) {
	int index = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
	if (index < V) {
		int vertex = vertices[index];
		int oldCommunity = deviceStructures.vertexCommunity[vertex];
		int newCommunity = deviceStructures.newVertexCommunity[vertex];
		if (oldCommunity != newCommunity) {
			deviceStructures.vertexCommunity[vertex] = newCommunity;
			atomicSub(&deviceStructures.communitySize[oldCommunity], 1);
			atomicAdd(&deviceStructures.communitySize[newCommunity], 1);
		}
	}
}

__global__ void updateOriginalToCommunity(device_structures deviceStructures) {
	int vertex = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
	if (vertex < *deviceStructures.originalV) {
		int community = deviceStructures.originalToCommunity[vertex];
		deviceStructures.originalToCommunity[vertex] = deviceStructures.vertexCommunity[community];
	}
}

void printVertexAssignments(host_structures& structures) {
//	for (int v = 0; v < structures.V; v++)
//		printf("%d ", structures.vertexCommunity[v]);
//	printf("\n");
	for (int c = 0; c < structures.V; c++) {
		printf("%d", c + 1);
		for (int v = 0; v < structures.V; v++)
			if (c == structures.vertexCommunity[v])
				printf(" %d", v + 1);
		printf("\n");
	}
}

struct isInBucket
{
	isInBucket(int llowerBound, int uupperBound, int *eedgesIndex) {
		lowerBound = llowerBound;
		upperBound = uupperBound;
		edgesIndex = eedgesIndex;
	}

	int lowerBound, upperBound;
	int *edgesIndex;
	__host__ __device__
	bool operator()(const int &v) const
	{
		int edgesNumber = edgesIndex[v + 1] - edgesIndex[v];
		return edgesNumber > lowerBound && edgesNumber <= upperBound;
	}
};

bool optimiseModularity(float minGain, device_structures& deviceStructures, host_structures& hostStructures) {
	int V = hostStructures.V;
	computeEdgesSum<<<blocksNumber(V, WARP_SIZE), dim3{WARP_SIZE, THREADS_PER_BLOCK / WARP_SIZE}>>>(deviceStructures);

	HANDLE_ERROR(hipMemcpy(hostStructures.edgesIndex, deviceStructures.edgesIndex,(V + 1) * (sizeof(int)), hipMemcpyDeviceToHost));

	int *partition = deviceStructures.partition;
	thrust::sequence(thrust::device, partition, partition + V, 0);

	float totalGain = minGain;
	bool wasAnythingChanged = false;
	while (totalGain >= minGain) {
		float modularityBefore = calculateModularity(V, deviceStructures);
		// TODO - separate case for last bucket
		for(int bucketNum= 0; bucketNum < bucketsSize - 2; bucketNum++) {
			dim3 blockDimension = dims[bucketNum];
			int vertexDegree = buckets[bucketNum + 1];
			int prime = primes[bucketNum];
			auto predicate = isInBucket(buckets[bucketNum], buckets[bucketNum + 1], hostStructures.edgesIndex);
			int *deviceVerticesEnd = thrust::partition(thrust::device, partition, partition + V, predicate);
			int verticesInBucket = thrust::distance(partition, deviceVerticesEnd);
			if (verticesInBucket > 0) {
				int sharedMemSize = blockDimension.y * prime * (sizeof(float) + sizeof(int));
//				if (blockDimension.x > WARP_SIZE)
//					sharedMemSize += 2 * THREADS_PER_BLOCK * sizeof(int);
				computeMove<<<blocksNumber(verticesInBucket, vertexDegree), blockDimension, sharedMemSize>>>(verticesInBucket, partition, prime,
						deviceStructures);
			}
			// updating vertex -> community assignment
			updateVertexCommunity<<<blocksNumber(V, 1), THREADS_PER_BLOCK>>>(verticesInBucket, partition, deviceStructures);
			// updating community weight
			thrust::fill(thrust::device, deviceStructures.communityWeight, deviceStructures.communityWeight + hostStructures.V, (float) 0);
			computeCommunityWeight<<<blocksNumber(V, 1), THREADS_PER_BLOCK>>>(deviceStructures);
		}

		float modularityAfter = calculateModularity(V, deviceStructures);
		totalGain = modularityAfter - modularityBefore;
		printf("before: %f, after: %f\n", modularityBefore, modularityAfter);
		wasAnythingChanged = wasAnythingChanged | (totalGain > 0);
	}
	HANDLE_ERROR(hipMemcpy(hostStructures.vertexCommunity, deviceStructures.vertexCommunity,
							hostStructures.V * sizeof(float), hipMemcpyDeviceToHost));
	V = hostStructures.V;
	printVertexAssignments(hostStructures);
	updateOriginalToCommunity<<<blocksNumber(hostStructures.originalV, 1), THREADS_PER_BLOCK>>>(deviceStructures);
	HANDLE_ERROR(hipFree(partition));
	return wasAnythingChanged;
}

__global__ void calculateModularityPerVertex(device_structures deviceStructures) {
	int community = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
	if (community < *deviceStructures.V) {
		float e_i_to_ci = 0;
		for (int vertex = 0; vertex < *deviceStructures.V; vertex++) {
			if (deviceStructures.vertexCommunity[vertex] == community) {
				for (int i = deviceStructures.edgesIndex[vertex]; i < deviceStructures.edgesIndex[vertex+1]; i++) {
					int neighbour = deviceStructures.edges[i];
					int neighbourCommunity = deviceStructures.vertexCommunity[neighbour];
					if (neighbourCommunity == community)
						e_i_to_ci += deviceStructures.weights[i];
				}
			}
		}
		float M = *deviceStructures.M;
		atomicAdd(deviceStructures.modularity, e_i_to_ci / (2 * M));
		float communityWeight = deviceStructures.communityWeight[community];
		atomicAdd(deviceStructures.modularity, -1 * communityWeight * communityWeight / (4 * M * M));
	}
}

float calculateModularity(int V, device_structures deviceStructures) {
	float modularity = 0;
	thrust::fill(thrust::device, deviceStructures.modularity, deviceStructures.modularity + 1, (float) 0);
	int blocksNumber = (V + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
	calculateModularityPerVertex<<<blocksNumber, THREADS_PER_BLOCK>>>(deviceStructures);
	HANDLE_ERROR(hipMemcpy(&modularity, deviceStructures.modularity, sizeof(float), hipMemcpyDeviceToHost));
	return modularity;
}

void initM(host_structures& hostStructures) {
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(M), &hostStructures.M, sizeof(float)));
}