#include "hip/hip_runtime.h"
#include "modularity_optimisation.cuh"
#include <thrust/partition.h>
#include <vector>

/**
 * Computes hashing (using double hashing) for open-addressing purposes of arrays in prepareHashArrays function.
 * @param val   value we want to insert
 * @param index current position
 * @param prime size of hash array
 * @return hash
 */
__device__ int getHash(int val, int index, int prime) {
	int h1 = val % prime;
	int h2 = 1 + (val % (prime - 1));
	return (h1 + index * h2) % prime;
}


/**
 * Computes sum of weights of edges adjacent to vertices (results are stored in vertexEdgesSum).
 * @param deviceStructures structures stored in device memory
 */
__global__ void computeEdgesSum(device_structures deviceStructures) {
	int verticesPerBlock = blockDim.y;
	int concurrentNeighbours = blockDim.x;
	float edgesSum = 0;
	int vertex = blockIdx.x * verticesPerBlock + threadIdx.y;
	if (vertex < *deviceStructures.V) {
		int startOffset = deviceStructures.edgesIndex[vertex], endOffset = deviceStructures.edgesIndex[vertex + 1];
		for (int index = startOffset + threadIdx.x; index < endOffset; index += concurrentNeighbours)
			edgesSum += deviceStructures.weights[index];

		for (int offset = concurrentNeighbours / 2; offset > 0; offset /= 2) {
			edgesSum += __shfl_down_sync(FULL_MASK, edgesSum, offset);
		}
		if (threadIdx.x == 0) {
			deviceStructures.vertexEdgesSum[vertex] = edgesSum;
		}
	}
}

/**
 * Computes sum of weights of edges adjacent to vertices (results are stored in vertexEdgesSum).
 * @param V               number of vertices
 * @param communityWeight community -> weight (sum of edges adjacent to vertices of community)
 * @param vertexCommunity vertex -> community assignment
 * @param vertexEdgesSum  vertex -> sum of edges adjacent to vertex
 */
__global__ void computeCommunityWeight(device_structures deviceStructures) {
	int vertex = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
	if (vertex < *deviceStructures.V) {
		int community = deviceStructures.vertexCommunity[vertex];
		atomicAdd(&deviceStructures.communityWeight[community], deviceStructures.vertexEdgesSum[vertex]);
	}
}

/**
 * Fills content of hashCommunity and hash_weights arrays that are later used in computeGain function.
 * @param community        neighbour's community
 * @param prime            prime number used for hashing
 * @param weight		   neighbour's weight
 * @param hashWeight	   table of sum of weights between vertices and communities
 * @param hashCommunity	   table informing which community's info is stored in given index
 * @param hashTablesOffset offset of the vertex in hash arrays (single hash array may contain multiple vertices)
 */
__device__ int prepareHashArrays(int community, int prime, float weight, float *hashWeight, int *hashCommunity,
								  int hashTablesOffset) {
	int it = 0, curPos;
	do {
		curPos = hashTablesOffset + getHash(community, it++, prime);
		if (hashCommunity[curPos] == community)
			atomicAdd(&hashWeight[curPos], weight);
		else if (hashCommunity[curPos] == -1) {
			if (atomicCAS(&hashCommunity[curPos], -1, community) == -1)
				atomicAdd(&hashWeight[curPos], weight);
			else if (hashCommunity[curPos] == community)
				atomicAdd(&hashWeight[curPos], weight);
		}
	} while (hashCommunity[curPos] != community);
	return curPos;
}

/**
 * Computes gain that would be obtained if we would move vertex to community.
 * @param vertex      	   vertex number
 * @param prime            prime number used for hashing (and size of vertex's area in hash arrays)
 * @param community 	   neighbour's community
 * @param currentCommunity current community of vertex
 * @param communityWeight  community -> weight (sum of edges adjacent to vertices of community)
 * @param vertexEdgesSum   vertex -> sum of edges adjacent to vertex
 * @param hashCommunity    table informing which community's info is stored in given index
 * @param hashWeight       table of sum of weights between vertices and communities
 * @param hashTablesOffset offset of the vertex in hash arrays (single hash array may contain multiple vertices
 * @return gain that would be obtained by moving vertex to community
 */
__device__ float computeGain(int vertex, int community, int currentCommunity, float *communityWeight,
							 float *vertexEdgesSum, float vertexToCommunity) {
	float communitySum = communityWeight[community];
	float currentCommunitySum = communityWeight[currentCommunity] - vertexEdgesSum[vertex];
	float gain = vertexToCommunity / M + vertexEdgesSum[vertex] * (currentCommunitySum - communitySum) / (2 * M * M);
	return gain;
}

/**
 * Finds new vertex -> community assignment (stored in newVertexCommunity) that maximise gains for each vertex.
 * @param V                number of vertices
 * @param vertices		   vertices
 * @param prime            prime number used for hashing
 * @param deviceStructures structures kept in device memory
 */
__device__ void computeMove(int V, int *vertices, int prime, device_structures deviceStructures, int *hashCommunity,
		float *hashWeight, float *vertexToCurrentCommunity, float *bestGains, int *bestCommunities) {
	int verticesPerBlock = blockDim.y;
	int vertexIndex = blockIdx.x * verticesPerBlock + threadIdx.y;
	if (vertexIndex < V) {
		int *vertexCommunity = deviceStructures.vertexCommunity, *edgesIndex = deviceStructures.edgesIndex,
		*edges = deviceStructures.edges, *communitySize = deviceStructures.communitySize,
		*newVertexCommunity = deviceStructures.newVertexCommunity;
		float *weights = deviceStructures.weights, *communityWeight = deviceStructures.communityWeight,
		*vertexEdgesSum = deviceStructures.vertexEdgesSum;

		int concurrentNeighbours = blockDim.x;
		int hashTablesOffset = threadIdx.y * prime;

        if (threadIdx.x == 0)
		    vertexToCurrentCommunity[threadIdx.y] = 0;
		for (unsigned int i = threadIdx.x; i < prime; i += concurrentNeighbours) {
			hashWeight[hashTablesOffset + i] = 0;
			hashCommunity[hashTablesOffset + i] = -1;
		}

		if (concurrentNeighbours > WARP_SIZE)
			__syncthreads();

		int vertex = vertices[vertexIndex];
		int currentCommunity = vertexCommunity[vertex];
		int bestCommunity = currentCommunity;
		float bestGain = 0;
		// putting data in hash table
		int neighbourIndex = threadIdx.x + edgesIndex[vertex];
		int upperBound = edgesIndex[vertex + 1];
		int curPos;

		while (neighbourIndex < upperBound) {
			int neighbour = edges[neighbourIndex];
			int community = vertexCommunity[neighbour];
			float weight = weights[neighbourIndex];
			// this lets us achieve ei -> C(i)\{i} instead of ei -> C(i)
			if (neighbour != vertex) {
				curPos = prepareHashArrays(community, prime, weight, hashWeight, hashCommunity, hashTablesOffset);
				if (community == currentCommunity)
					 atomicAdd(&vertexToCurrentCommunity[threadIdx.y], weight);
			}
			if ((community < currentCommunity || communitySize[community] > 1 || communitySize[currentCommunity] > 1) &&
				community != currentCommunity) {
				float gain = computeGain(vertex, community, currentCommunity, communityWeight, vertexEdgesSum, hashWeight[curPos]);
				if (gain > bestGain || (gain == bestGain && community < bestCommunity)) {
					bestGain = gain;
					bestCommunity = community;
				}
			}
			neighbourIndex += concurrentNeighbours;
		}

		if (concurrentNeighbours <= WARP_SIZE) {
			for (unsigned int offset = concurrentNeighbours / 2; offset > 0; offset /= 2) {
				float otherGain = __shfl_down_sync(FULL_MASK, bestGain, offset);
				int otherCommunity = __shfl_down_sync(FULL_MASK, bestCommunity, offset);
				if (otherGain > bestGain || (otherGain == bestGain && otherCommunity < bestCommunity)) {
					bestGain = otherGain;
					bestCommunity = otherCommunity;
				}
			}
		} else {
            bestGains[threadIdx.x] = bestGain;
            bestCommunities[threadIdx.x] = bestCommunity;
			for (unsigned int offset = concurrentNeighbours / 2; offset > 0; offset /= 2) {
				__syncthreads();
				if (threadIdx.x < offset) {
					float otherGain = bestGains[threadIdx.x + offset];
					int otherCommunity = bestCommunities[threadIdx.x + offset];
					if (otherGain > bestGains[threadIdx.x] ||
					   (otherGain == bestGains[threadIdx.x] && otherCommunity < bestCommunities[threadIdx.x])) {
						bestGains[threadIdx.x] = otherGain;
						bestCommunities[threadIdx.x] = otherCommunity;
					}
				}
			}
            bestGain = bestGains[threadIdx.x];
            bestCommunity = bestCommunities[threadIdx.x];
		}
		if (threadIdx.x == 0 && bestGain - vertexToCurrentCommunity[threadIdx.y] / M > 0) {
			newVertexCommunity[vertex] = bestCommunity;
		} else {
			newVertexCommunity[vertex] = currentCommunity;
		}
	}
}

__global__ void computeMoveShared(int V, int *vertices, int prime, device_structures deviceStructures) {
	int verticesPerBlock = blockDim.y;
	int vertexIndex = blockIdx.x * verticesPerBlock + threadIdx.y;
	if (vertexIndex < V) {
		extern __shared__ int s[];
		int *hashCommunity = s;
		auto *hashWeight = (float *) &hashCommunity[verticesPerBlock * prime];
		auto *vertexToCurrentCommunity = (float *) &hashWeight[verticesPerBlock * prime];
		float *bestGains = &vertexToCurrentCommunity[verticesPerBlock];
		int *bestCommunities = (int *) &bestGains[THREADS_PER_BLOCK];
		computeMove(V, vertices, prime, deviceStructures, hashCommunity, hashWeight, vertexToCurrentCommunity,
				bestGains, bestCommunities);
	}
}

__global__ void computeMoveGlobal(int V, int *vertices, int prime, device_structures deviceStructures, int *hashCommunity, float *hashWeight) {
	int verticesPerBlock = blockDim.y;
	int vertexIndex = blockIdx.x * verticesPerBlock + threadIdx.y;
	if (vertexIndex < V) {
		extern __shared__ int s[];
		auto *vertexToCurrentCommunity = (float *) s;
		float *bestGains = &vertexToCurrentCommunity[verticesPerBlock];
		int *bestCommunities = (int *) &bestGains[THREADS_PER_BLOCK];
		hashCommunity = hashCommunity + blockIdx.x * prime;
		hashWeight = hashWeight + blockIdx.x * prime;
		computeMove(V, vertices, prime, deviceStructures, hashCommunity, hashWeight, vertexToCurrentCommunity,
					bestGains, bestCommunities);
	}
}

/**
 * Updates vertexCommunity content based on newVertexCommunity content..
 * Additionally, updates communitySize.
 * @param V                number of vertices
 * @param vertices         vertices
 * @param deviceStructures structures kept in device memory
 */
__global__ void updateVertexCommunity(int V, int *vertices, device_structures deviceStructures) {
	int index = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
	if (index < V) {
		int vertex = vertices[index];
		int oldCommunity = deviceStructures.vertexCommunity[vertex];
		int newCommunity = deviceStructures.newVertexCommunity[vertex];
		if (oldCommunity != newCommunity) {
			deviceStructures.vertexCommunity[vertex] = newCommunity;
			atomicSub(&deviceStructures.communitySize[oldCommunity], 1);
			atomicAdd(&deviceStructures.communitySize[newCommunity], 1);
		}
	}
}

__global__ void updateOriginalToCommunity(device_structures deviceStructures) {
	int vertex = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
	if (vertex < *deviceStructures.originalV) {
		int community = deviceStructures.originalToCommunity[vertex];
		deviceStructures.originalToCommunity[vertex] = deviceStructures.vertexCommunity[community];
	}
}

struct isInBucket
{
	isInBucket(int llowerBound, int uupperBound, int *eedgesIndex) {
		lowerBound = llowerBound;
		upperBound = uupperBound;
		edgesIndex = eedgesIndex;
	}

	int lowerBound, upperBound;
	int *edgesIndex;
	__host__ __device__
	bool operator()(const int &v) const
	{
		int edgesNumber = edgesIndex[v + 1] - edgesIndex[v];
		return edgesNumber > lowerBound && edgesNumber <= upperBound;
	}
};

int getMaxDegree(host_structures& hostStructures) {
    int curMax = 0;
    for (int i = 0; i < hostStructures.V; i++)
        curMax = std::max(curMax, hostStructures.edgesIndex[i+1] - hostStructures.edgesIndex[i]);
    return curMax;
}

bool optimiseModularity(float minGain, device_structures& deviceStructures, host_structures& hostStructures) {
	int V = hostStructures.V;
	computeEdgesSum<<<blocksNumber(V, WARP_SIZE), dim3{WARP_SIZE, THREADS_PER_BLOCK / WARP_SIZE}>>>(deviceStructures);
    HANDLE_ERROR(hipMemcpy(hostStructures.edgesIndex, deviceStructures.edgesIndex,(V + 1) * (sizeof(int)), hipMemcpyDeviceToHost));

	int *partition = deviceStructures.partition;
	thrust::sequence(thrust::device, partition, partition + V, 0);

	int lastBucketPrime = getPrime(getMaxDegree(hostStructures) * 1.5);
	int *hashCommunity;
	float *hashWeight;
    int lastBucketNum = bucketsSize - 2;
    dim3 lastBlockDimension = dims[lastBucketNum];
    auto predicate = isInBucket(buckets[lastBucketNum], buckets[lastBucketNum + 1], hostStructures.edgesIndex);
    int *deviceVerticesEnd = thrust::partition(thrust::device, partition, partition + V, predicate);
    int verticesInLastBucket = thrust::distance(partition, deviceVerticesEnd);
    if (verticesInLastBucket > 0) {
        unsigned int blocksNum = (verticesInLastBucket + lastBlockDimension.y - 1) / lastBlockDimension.y;
        HANDLE_ERROR(hipMalloc((void**)&hashCommunity, lastBucketPrime * blocksNum	* sizeof(int)));
        HANDLE_ERROR(hipMalloc((void**)&hashWeight, lastBucketPrime * blocksNum * sizeof(float)));
    }

	float totalGain = minGain;
	bool wasAnythingChanged = false;
	while (totalGain >= minGain) {
		float modularityBefore = calculateModularity(V, hostStructures.M, deviceStructures);
		for(int bucketNum= 0; bucketNum < bucketsSize - 2; bucketNum++) {
			dim3 blockDimension = dims[bucketNum];
			int prime = primes[bucketNum];
			auto predicate = isInBucket(buckets[bucketNum], buckets[bucketNum + 1], hostStructures.edgesIndex);
			deviceVerticesEnd = thrust::partition(thrust::device, partition, partition + V, predicate);
			int verticesInBucket = thrust::distance(partition, deviceVerticesEnd);
			if (verticesInBucket > 0) {
                int sharedMemSize =
                        blockDimension.y * prime * (sizeof(float) + sizeof(int)) + blockDimension.y * sizeof(float);
                if (blockDimension.x > WARP_SIZE)
                    sharedMemSize += THREADS_PER_BLOCK * (sizeof(int) + sizeof(float));
                int blocksNum = (verticesInBucket + blockDimension.y - 1) / blockDimension.y;
                computeMoveShared<<<blocksNum, blockDimension, sharedMemSize>>>(verticesInBucket, partition, prime,
                                                                                   deviceStructures);
                // updating vertex -> community assignment
                updateVertexCommunity<<<blocksNumber(V, 1), THREADS_PER_BLOCK>>>(verticesInBucket, partition,
                                                                                 deviceStructures);
                // updating community weight
                thrust::fill(thrust::device, deviceStructures.communityWeight,
                             deviceStructures.communityWeight + hostStructures.V, (float) 0);
                computeCommunityWeight<<<blocksNumber(V, 1), THREADS_PER_BLOCK>>>(deviceStructures);
            }
		}

		// last bucket case
		deviceVerticesEnd = thrust::partition(thrust::device, partition, partition + V, predicate);
		int verticesInBucket = thrust::distance(partition, deviceVerticesEnd);
		if (verticesInBucket > 0) {
			unsigned int blocksNum = (verticesInBucket + lastBlockDimension.y - 1) / lastBlockDimension.y;
			int sharedMemSize = THREADS_PER_BLOCK * (sizeof(int) + sizeof(float)) + lastBlockDimension.y * sizeof(float);
			computeMoveGlobal<<<blocksNum, lastBlockDimension, sharedMemSize>>>(
					verticesInBucket, partition, lastBucketPrime,deviceStructures, hashCommunity, hashWeight);
		}
        // updating vertex -> community assignment
        updateVertexCommunity<<<blocksNumber(V, 1), THREADS_PER_BLOCK>>>(verticesInBucket, partition,
                                                                         deviceStructures);
        // updating community weight
        thrust::fill(thrust::device, deviceStructures.communityWeight,
                     deviceStructures.communityWeight + hostStructures.V, (float) 0);
        computeCommunityWeight<<<blocksNumber(V, 1), THREADS_PER_BLOCK>>>(deviceStructures);

		float modularityAfter = calculateModularity(V, hostStructures.M, deviceStructures);
		totalGain = modularityAfter - modularityBefore;
		wasAnythingChanged = wasAnythingChanged | (totalGain > 0);
	}
	HANDLE_ERROR(hipMemcpy(hostStructures.vertexCommunity, deviceStructures.vertexCommunity,
							hostStructures.V * sizeof(float), hipMemcpyDeviceToHost));
	if (verticesInLastBucket) {
        HANDLE_ERROR(hipFree(hashCommunity));
        HANDLE_ERROR(hipFree(hashWeight));
    }
	updateOriginalToCommunity<<<blocksNumber(hostStructures.originalV, 1), THREADS_PER_BLOCK>>>(deviceStructures);
	return wasAnythingChanged;
}

__global__ void calculateToOwnCommunity(device_structures deviceStructures) {
    int verticesPerBlock = blockDim.y;
    int concurrentNeighbours = blockDim.x;
    float edgesSum = 0;
    int vertex = blockIdx.x * verticesPerBlock + threadIdx.y;
    int community = deviceStructures.vertexCommunity[vertex];
    if (vertex < *deviceStructures.V) {
        int startOffset = deviceStructures.edgesIndex[vertex], endOffset = deviceStructures.edgesIndex[vertex + 1];
        for (int index = startOffset + threadIdx.x; index < endOffset; index += concurrentNeighbours) {
            int neighbour = deviceStructures.edges[index];
            if (deviceStructures.vertexCommunity[neighbour] == community)
                edgesSum += deviceStructures.weights[index];
        }

        for (unsigned int offset = concurrentNeighbours / 2; offset > 0; offset /= 2) {
            edgesSum += __shfl_down_sync(FULL_MASK, edgesSum, offset);
        }
        if (threadIdx.x == 0) {
            deviceStructures.toOwnCommunity[vertex] = edgesSum;
        }
    }
}

struct square {
    __device__ float operator()(const float &x) const {
        return x * x;
    }
};


float calculateModularity(int V, float M, device_structures deviceStructures) {
    calculateToOwnCommunity<<<blocksNumber(V, WARP_SIZE), dim3{WARP_SIZE, THREADS_PER_BLOCK / WARP_SIZE}>>>(deviceStructures);
    float communityWeightSum = thrust::transform_reduce(thrust::device, deviceStructures.communityWeight,
            deviceStructures.communityWeight + V, square(), 0.0, thrust::plus<float>());
    float toOwnCommunity = thrust::reduce(thrust::device, deviceStructures.toOwnCommunity, deviceStructures.toOwnCommunity + V);
	return toOwnCommunity / (2 * M) - communityWeightSum  / (4 * M * M);
}

void printOriginalToCommunity(device_structures& deviceStructures, host_structures& hostStructures) {
	std::vector<int> communityToVector[hostStructures.V];
	HANDLE_ERROR(hipMemcpy(hostStructures.originalToCommunity, deviceStructures.originalToCommunity,
			hostStructures.originalV * sizeof(int), hipMemcpyDeviceToHost));
	for (int vector = 0; vector < hostStructures.originalV; vector++) {
		int community = hostStructures.originalToCommunity[vector];
		communityToVector[community].emplace_back(vector);
	}
	printf("%d\n", hostStructures.V);
	for (int community = 0; community < hostStructures.V; community++) {
		printf("%d", community + 1);
		for (int i = 0; i < communityToVector[community].size(); i++)
			printf(" %d", communityToVector[community][i] + 1);
		printf("\n");
	}
}

void initM(host_structures& hostStructures) {
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(M), &hostStructures.M, sizeof(float)));
}