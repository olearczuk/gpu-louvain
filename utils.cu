#include "hip/hip_runtime.h"
#include "utils.cuh"
#include <vector>
#include <iostream>
#include <thrust/partition.h>
#include <fstream>
#include <getopt.h>
#include <sstream>

host_structures readInputData(char *fileName) {
	std::fstream file;
	file.open(fileName);
    int V, E;
	std::string s;
	do {
		std::getline(file, s);
	} while (s[0] == '%');
	std::istringstream stream(s);
    stream >> V >> V >> E;
    int v1, v2;
    float w;
    host_structures hostStructures;
	hostStructures.originalV = V;
	hostStructures.V = V;
    HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.vertexCommunity, V * sizeof(int), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.communityWeight, V * sizeof(float), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.edgesIndex, (V + 1) * sizeof(int), hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.originalToCommunity, V * sizeof(int), hipHostMallocDefault));

    std::vector<std::vector<std::pair<int, float>>> neighbours(V);
    // TODO: here is assumption that graph is undirected
    int aux = E;
    for (int i = 0; i < aux; i++) {
        file >> v1 >> v2 >> w;
        v1--;
        v2--;
		hostStructures.communityWeight[v1] += w;
        neighbours[v1].emplace_back(v2, w);
        if (v1 != v2) {
            E++;
			hostStructures.communityWeight[v2] += w;
            neighbours[v2].emplace_back(v1, w);
			hostStructures.M += w;
        }
		hostStructures.M += w;
    }
    hostStructures.M /= 2;
    HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.edges, E * sizeof(int), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.weights, E * sizeof(float), hipHostMallocDefault));
	hostStructures.E = E;
    int index = 0;
    for (int v = 0; v < V; v++) {
		hostStructures.edgesIndex[v] = index;
        for (auto & it : neighbours[v]) {
			hostStructures.edges[index] = it.first;
			hostStructures.weights[index] = it.second;
            index++;
        }
    }
	hostStructures.edgesIndex[V] = E;
    file.close();
    return hostStructures;
}

void copyStructures(host_structures& hostStructures, device_structures& deviceStructures,
					aggregation_phase_structures& aggregationPhaseStructures) {
	// copying from deviceStructures to hostStructures
	int V = hostStructures.V, E = hostStructures.E;
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.vertexCommunity, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.communityWeight, V * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.edges, E * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.weights, E * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.edgesIndex, (V + 1) * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.originalToCommunity, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.vertexEdgesSum, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.newVertexCommunity, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.V, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.E, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.originalV, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.communitySize, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.partition, V * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&deviceStructures.toOwnCommunity, V * sizeof(int)));


	thrust::fill(thrust::device, deviceStructures.communitySize, deviceStructures.communitySize + V, 1);
	thrust::sequence(thrust::device, deviceStructures.vertexCommunity, deviceStructures.vertexCommunity + V, 0);
	thrust::sequence(thrust::device, deviceStructures.newVertexCommunity, deviceStructures.newVertexCommunity + V, 0);
	thrust::sequence(thrust::device, deviceStructures.originalToCommunity, deviceStructures.originalToCommunity + V, 0);

	HANDLE_ERROR(hipMemcpy(deviceStructures.communityWeight, hostStructures.communityWeight, V * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.edges, hostStructures.edges, E * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.weights, hostStructures.weights, E * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.edgesIndex, hostStructures.edgesIndex, (V + 1) * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.V, &hostStructures.V, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.E, &hostStructures.E, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.originalV, &hostStructures.originalV, sizeof(int), hipMemcpyHostToDevice));

	// preparing aggregationPhaseStructures
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.communityDegree, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.newID, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.edgePos, V * sizeof(int)));;
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.vertexStart, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.orderedVertices, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.edgeIndexToCurPos, E * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.newEdges, E * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.newWeights, E * sizeof(float)));
}

void deleteStructures(host_structures& hostStructures, device_structures& deviceStructures,
					  aggregation_phase_structures& aggregationPhaseStructures) {
    HANDLE_ERROR(hipHostFree(hostStructures.vertexCommunity));
    HANDLE_ERROR(hipHostFree(hostStructures.communityWeight));
    HANDLE_ERROR(hipHostFree(hostStructures.edges));
    HANDLE_ERROR(hipHostFree(hostStructures.weights));
    HANDLE_ERROR(hipHostFree(hostStructures.edgesIndex));
    HANDLE_ERROR(hipHostFree(hostStructures.originalToCommunity));


	HANDLE_ERROR(hipFree(deviceStructures.originalV));
    HANDLE_ERROR(hipFree(deviceStructures.vertexCommunity));
	HANDLE_ERROR(hipFree(deviceStructures.communityWeight));
	HANDLE_ERROR(hipFree(deviceStructures.edges));
	HANDLE_ERROR(hipFree(deviceStructures.weights));
	HANDLE_ERROR(hipFree(deviceStructures.edgesIndex));
	HANDLE_ERROR(hipFree(deviceStructures.originalToCommunity));
	HANDLE_ERROR(hipFree(deviceStructures.vertexEdgesSum));
	HANDLE_ERROR(hipFree(deviceStructures.newVertexCommunity));
	HANDLE_ERROR(hipFree(deviceStructures.E));
	HANDLE_ERROR(hipFree(deviceStructures.V));
	HANDLE_ERROR(hipFree(deviceStructures.communitySize));
	HANDLE_ERROR(hipFree(deviceStructures.partition));
    HANDLE_ERROR(hipFree(deviceStructures.toOwnCommunity));

	HANDLE_ERROR(hipFree(aggregationPhaseStructures.communityDegree));
	HANDLE_ERROR(hipFree(aggregationPhaseStructures.newID));
	HANDLE_ERROR(hipFree(aggregationPhaseStructures.edgePos));
	HANDLE_ERROR(hipFree(aggregationPhaseStructures.vertexStart));
	HANDLE_ERROR(hipFree(aggregationPhaseStructures.orderedVertices));
	HANDLE_ERROR(hipFree(aggregationPhaseStructures.edgeIndexToCurPos));
	HANDLE_ERROR(hipFree(aggregationPhaseStructures.newEdges));
	HANDLE_ERROR(hipFree(aggregationPhaseStructures.newWeights));
}

int blocksNumber(int V, int threadsPerVertex) {
	return (V * threadsPerVertex + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
}

bool isPrime(int n) {
	for (int i = 2; i < sqrt(n) + 1; i++)
		if (n % i == 0)
			return false;
	return true;
}

int getPrime(int n) {
	do {
		n++;
	} while(!isPrime(n));
	return n;
}

void parseCommandLineArgs(int argc, char *argv[], float *minGain, bool *isVerbose, char **fileName) {
	bool isF, isG;
	char opt;
	while ((opt = getopt(argc, argv, "f:g:v")) != -1) {
		switch (opt) {
			case 'g':
				isG = true;
				*minGain = strtof(optarg, NULL);
				break;
			case 'v':
				*isVerbose = true;
				break;
			case 'f':
				isF = true;
				*fileName = optarg;
				break;
			default:
				printf("Usage: ./gpulouvain -f mtx-matrix-file -g min-gain [-v]\n");
				exit(1);
		}
	}
	if (!isF || !isG) {
		printf("Usage: ./gpulouvain -f mtx-matrix-file -g min-gain [-v]\n");
		exit(1);
	}
}