#include "utils.cuh"
#include <vector>
#include <iostream>

host_structures readInputData() {
    int V, E;
    std::cin >> V >> V >> E;
    int v1, v2;
    float w;
    host_structures hostStructures;
	hostStructures.originalV = V;
	hostStructures.V = V;
    HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.vertexCommunity, V * sizeof(int), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.communityWeight, V * sizeof(float), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.edgesIndex, (V + 1) * sizeof(int), hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.originalToCommunity, V * sizeof(int), hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.vertices, V * sizeof(int), hipHostMallocDefault));

    std::vector<std::pair<int, float>> neighbours[V];
    for (int v = 0; v < V; v++) {
		hostStructures.vertexCommunity[v] = v;
		hostStructures.originalToCommunity[v] = v;
		hostStructures.vertices[v] = v;
    }
    // TODO: here is assumption that graph is undirected
    int aux = E;
    for (int i = 0; i < aux; i++) {
        std::cin >> v1 >> v2 >> w;
        v1--;
        v2--;
		hostStructures.communityWeight[v1] += w;
        neighbours[v1].emplace_back(v2, w);
        if (v1 != v2) {
            E++;
			hostStructures.communityWeight[v2] += w;
            neighbours[v2].emplace_back(v1, w);
        }
		hostStructures.M += w;
    }
    HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.edges, E * sizeof(int), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.weights, E * sizeof(float), hipHostMallocDefault));
	hostStructures.E = E;
    int index = 0;
    for (int v = 0; v < V; v++) {
		hostStructures.edgesIndex[v] = index;
        for (auto & it : neighbours[v]) {
			hostStructures.edges[index] = it.first;
			hostStructures.weights[index] = it.second;
            index++;
        }
    }
	hostStructures.edgesIndex[V] = E;
    return hostStructures;
}

void copyStructures(host_structures& hostStructures, device_structures& deviceStructures) {
	int V = hostStructures.V, E = hostStructures.E;
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.vertexCommunity, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.communityWeight, V * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.edges, E * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.weights, E * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.edgesIndex, (V + 1) * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.originalToCommunity, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.vertexEdgesSum, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.newVertexCommunity, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.vertices, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.V, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.E, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.originalV, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.totalGain, sizeof(float)));

	HANDLE_ERROR(hipMemcpy(deviceStructures.vertexCommunity, hostStructures.vertexCommunity, V * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.newVertexCommunity, hostStructures.vertexCommunity, V * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.communityWeight, hostStructures.communityWeight, V * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.edges, hostStructures.edges, E * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.weights, hostStructures.weights, E * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.edgesIndex, hostStructures.edgesIndex, (V + 1) * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.originalToCommunity, hostStructures.originalToCommunity, V * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.vertices, hostStructures.vertices, V * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.V, &hostStructures.V, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.E, &hostStructures.E, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.originalV, &hostStructures.originalV, sizeof(int), hipMemcpyHostToDevice));
}

void deleteStructures(host_structures& hostStructures, device_structures& deviceStructures) {
    HANDLE_ERROR(hipHostFree(hostStructures.vertexCommunity));
    HANDLE_ERROR(hipHostFree(hostStructures.communityWeight));
    HANDLE_ERROR(hipHostFree(hostStructures.edges));
    HANDLE_ERROR(hipHostFree(hostStructures.weights));
    HANDLE_ERROR(hipHostFree(hostStructures.edgesIndex));
    HANDLE_ERROR(hipHostFree(hostStructures.originalToCommunity));

	HANDLE_ERROR(hipFree(deviceStructures.V));
	HANDLE_ERROR(hipFree(deviceStructures.originalV));
    HANDLE_ERROR(hipFree(deviceStructures.vertexCommunity));
	HANDLE_ERROR(hipFree(deviceStructures.communityWeight));
	HANDLE_ERROR(hipFree(deviceStructures.edges));
	HANDLE_ERROR(hipFree(deviceStructures.weights));
	HANDLE_ERROR(hipFree(deviceStructures.edgesIndex));
	HANDLE_ERROR(hipFree(deviceStructures.originalToCommunity));
	HANDLE_ERROR(hipFree(deviceStructures.vertexEdgesSum));
	HANDLE_ERROR(hipFree(deviceStructures.newVertexCommunity));
	HANDLE_ERROR(hipFree(deviceStructures.vertices));
	HANDLE_ERROR(hipFree(deviceStructures.totalGain));
	HANDLE_ERROR(hipFree(deviceStructures.E));
}
