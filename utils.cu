#include "utils.cuh"
#include <vector>
#include <iostream>
#include <thrust/partition.h>
#include <fstream>

host_structures readInputData(char *fileName) {
	std::fstream file;
	file.open(fileName);
    int V, E;
    file >> V >> V >> E;
    int v1, v2;
    float w;
    host_structures hostStructures;
	hostStructures.originalV = V;
	hostStructures.V = V;
    HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.vertexCommunity, V * sizeof(int), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.communityWeight, V * sizeof(float), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.edgesIndex, (V + 1) * sizeof(int), hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.originalToCommunity, V * sizeof(int), hipHostMallocDefault));

    std::vector<std::pair<int, float>> neighbours[V];
    for (int v = 0; v < V; v++) {
		hostStructures.vertexCommunity[v] = v;
		hostStructures.originalToCommunity[v] = v;
    }
    // TODO: here is assumption that graph is undirected
    int aux = E;
    for (int i = 0; i < aux; i++) {
        file >> v1 >> v2 >> w;
        v1--;
        v2--;
		hostStructures.communityWeight[v1] += w;
        neighbours[v1].emplace_back(v2, w);
        if (v1 != v2) {
            E++;
			hostStructures.communityWeight[v2] += w;
            neighbours[v2].emplace_back(v1, w);
			hostStructures.M += w;
        }
		hostStructures.M += w;
    }
    HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.edges, E * sizeof(int), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.weights, E * sizeof(float), hipHostMallocDefault));
	hostStructures.E = E;
    int index = 0;
    for (int v = 0; v < V; v++) {
		hostStructures.edgesIndex[v] = index;
        for (auto & it : neighbours[v]) {
			hostStructures.edges[index] = it.first;
			hostStructures.weights[index] = it.second;
            index++;
        }
    }
	hostStructures.edgesIndex[V] = E;
    file.close();
    return hostStructures;
}

void copyStructures(host_structures& hostStructures, device_structures& deviceStructures) {
	int V = hostStructures.V, E = hostStructures.E;
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.vertexCommunity, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.communityWeight, V * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.edges, E * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.weights, E * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.edgesIndex, (V + 1) * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.originalToCommunity, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.vertexEdgesSum, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.newVertexCommunity, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.V, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.E, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.originalV, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.communitySize, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.partition, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.M, sizeof(float)));

	thrust::fill(thrust::device, deviceStructures.communitySize, deviceStructures.communitySize + hostStructures.V, 1);

	HANDLE_ERROR(hipMemcpy(deviceStructures.vertexCommunity, hostStructures.vertexCommunity, V * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.newVertexCommunity, hostStructures.vertexCommunity, V * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.communityWeight, hostStructures.communityWeight, V * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.edges, hostStructures.edges, E * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.weights, hostStructures.weights, E * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.edgesIndex, hostStructures.edgesIndex, (V + 1) * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.originalToCommunity, hostStructures.originalToCommunity, V * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.V, &hostStructures.V, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.E, &hostStructures.E, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.originalV, &hostStructures.originalV, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.M, &hostStructures.M, sizeof(float), hipMemcpyHostToDevice));
}

void deleteStructures(host_structures& hostStructures, device_structures& deviceStructures) {
    HANDLE_ERROR(hipHostFree(hostStructures.vertexCommunity));
    HANDLE_ERROR(hipHostFree(hostStructures.communityWeight));
    HANDLE_ERROR(hipHostFree(hostStructures.edges));
    HANDLE_ERROR(hipHostFree(hostStructures.weights));
    HANDLE_ERROR(hipHostFree(hostStructures.edgesIndex));
    HANDLE_ERROR(hipHostFree(hostStructures.originalToCommunity));

	HANDLE_ERROR(hipFree(deviceStructures.V));
	HANDLE_ERROR(hipFree(deviceStructures.originalV));
    HANDLE_ERROR(hipFree(deviceStructures.vertexCommunity));
	HANDLE_ERROR(hipFree(deviceStructures.communityWeight));
	HANDLE_ERROR(hipFree(deviceStructures.edges));
	HANDLE_ERROR(hipFree(deviceStructures.weights));
	HANDLE_ERROR(hipFree(deviceStructures.edgesIndex));
	HANDLE_ERROR(hipFree(deviceStructures.originalToCommunity));
	HANDLE_ERROR(hipFree(deviceStructures.vertexEdgesSum));
	HANDLE_ERROR(hipFree(deviceStructures.newVertexCommunity));
	HANDLE_ERROR(hipFree(deviceStructures.M));
	HANDLE_ERROR(hipFree(deviceStructures.E));
}

int blocksNumber(int V, int threadsPerVertex) {
	return (V * threadsPerVertex + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
}