#include "utils.cuh"
#include "modularity_optimisation.cuh"
#include "community_aggregation.cuh"


int main(int argc, char *argv[]) {
	char *fileName;
	float minGain;
	bool isVerbose;
	parseCommandLineArgs(argc, argv, &minGain, &isVerbose, &fileName);

    auto hostStructures = readInputData(fileName);
    device_structures deviceStructures;
    aggregation_phase_structures aggregationPhaseStructures;

    hipEvent_t start, stop;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, 0));
	copyStructures(hostStructures, deviceStructures, aggregationPhaseStructures);
	initM(hostStructures);
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	float memoryTime;
	HANDLE_ERROR(hipEventElapsedTime(&memoryTime, start, stop));

	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, 0));
	for (;;) {
		if (!optimiseModularity(minGain, deviceStructures, hostStructures))
			break;
		aggregateCommunities(deviceStructures, hostStructures, aggregationPhaseStructures);
	}
	int V;
	HANDLE_ERROR(hipMemcpy(&V, deviceStructures.V, sizeof(int), hipMemcpyDeviceToHost));
	printf("%f\n", calculateModularity(V, hostStructures.M, deviceStructures));
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	float algorithmTime;
	HANDLE_ERROR(hipEventElapsedTime(&algorithmTime, start, stop));
	printf("%f %f\n", algorithmTime, algorithmTime + memoryTime);
	if (isVerbose)
		printOriginalToCommunity(deviceStructures, hostStructures);
	deleteStructures(hostStructures, deviceStructures, aggregationPhaseStructures);
}
